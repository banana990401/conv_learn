#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "../../include/verify.h"
#include "../../include/conv_2d.h"

__global__ void implgemm(param_t param)
{
    uint32_t tx = threadIdx.x;
    int bx      = blockIdx.x;
    int by      = blockIdx.y;

    // warp tile
    const uint32_t lane_id   = tx % 32;
    const uint32_t warp_id   = tx / 32;
    const uint32_t mma_tid_x = lane_id % 8;
    const uint32_t mma_tid_y = lane_id / 8;
    // lds addr
    uint32_t weight_lds_addr = (warp_id / 2) * 16 + mma_tid_y * 4;
    uint32_t input_lds_addr  = (warp_id % 2) * 32 + mma_tid_x * 4;

    int x = bx * 64 + input_lds_addr;
    int y = by * 64 + weight_lds_addr;
    int z = blockIdx.z;

    __shared__ float smem_input[4 * 64];
    __shared__ float smem_weight[4 * 64];

    int pos_oh = (bx * 64 + tx % 64) / param.Ow;
    int pos_ow = (bx * 64 + tx % 64) % param.Ow;

    int pos_h = pos_oh * param.u - param.p;
    int pos_w = pos_ow * param.v - param.q;

    int in_offset     = z * param.c * param.h * param.w;
    int weight_offset = (by * 64 + tx / 4) * param.c * param.r * param.s;

    int in_channel_offset     = param.h * param.w;

    // sts addr(ld：取数据单元，st：存数据单元)
    uint32_t weight_sts_addr = (tx % 4) * 64 + (tx / 4);
    uint32_t input_sts_addr  = (tx / 64) * 64 + (tx % 64);

    float output_frag[4][4];
#pragma unroll
    for(int i = 0; i < 4; i++)
    {
#pragma unroll
        for(int j = 0; j < 4; j++)
        {
            output_frag[i][j] = 0.0f;
        }
    }

    for(int crs = 0; crs < param.c * param.r * param.s; crs += 4)
    {
        int weight_offset_tmp        = crs + tx % 4;
        smem_weight[weight_sts_addr] = param.weight[weight_offset + weight_offset_tmp];

        int cur_c         = (crs + tx / 64) / (param.r * param.s);
        int cur_r         = ((crs + tx / 64) % (param.r * param.s)) / param.s;
        int cur_s         = ((crs + tx / 64) % (param.r * param.s)) % param.s;
        int cur_h         = pos_h + cur_r;
        int cur_w         = pos_w + cur_s;
        int in_offset_tmp = cur_c * in_channel_offset + cur_h * param.w + cur_w;
        if(cur_h >= 0 && cur_w >= 0 && cur_h < param.h && cur_w < param.w)
        {
            smem_input[input_sts_addr] = param.input[in_offset + in_offset_tmp];
        }
        else
        {
            smem_input[input_sts_addr] = 0.0f;
        }
        __syncthreads();

#pragma unroll
        for(int i = 0; i < 4; i++)
        {
#pragma unroll
            for(int j = 0; j < 4; j++)
            {
#pragma unroll
                for(int subcrs = 0; subcrs < 4; subcrs++)
                {
                    output_frag[i][j] += smem_weight[weight_lds_addr + subcrs * 64 + i] *
                                         smem_input[input_lds_addr + subcrs * 64 + j];
                }
            }
        }
        __syncthreads();
    }

    int out_offset;
#pragma unroll
    for(int i = 0; i < 4; i++)
    {
#pragma unroll
        for(int j = 0; j < 4; j++)
        {
            out_offset = z * param.k * param.Oh * param.Ow + (y + i) * param.Oh * param.Ow + x + j;
            if(x + j < param.Oh * param.Ow && y + i < param.k)
            {
                param.output[out_offset] = output_frag[i][j];
            }
        }
    }
}

void launch_implgemm(param_t param)
{
    unsigned int n = param.n;
    unsigned int h = param.h;
    unsigned int w = param.w;
    unsigned int k = param.k;
    unsigned int r = param.r;
    unsigned int s = param.s;
    unsigned int u = param.u;
    unsigned int v = param.v;
    unsigned int p = param.p;
    unsigned int q = param.q;

    int out_h = (h - r + 2 * p) / u + 1;
    int out_w = (w - s + 2 * q) / v + 1;

    param.Oh = out_h;
    param.Ow = out_w;

    int block_x  = ((out_h * out_w + 63) / 64);
    int block_y  = (k + 63) / 64;
    int block_z  = n;
    int thread_x = 256;
    int thread_y = 1;
    int thread_z = 1;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(thread_x, thread_y, thread_z);

    implgemm<<<grid, block>>>(param);
}

int main(int argc, char** argv)
{
    unsigned int n = atoi(argv[1]);
    unsigned int c = atoi(argv[2]);
    unsigned int h = atoi(argv[3]);
    unsigned int w = atoi(argv[4]);
    unsigned int k = atoi(argv[5]);
    unsigned int r = atoi(argv[6]);
    unsigned int s = atoi(argv[7]);
    unsigned int u = atoi(argv[8]);
    unsigned int v = atoi(argv[9]);
    unsigned int p = atoi(argv[10]);
    unsigned int q = atoi(argv[11]);

    int out_h = (h - r + 2 * p) / u + 1;
    int out_w = (w - s + 2 * q) / v + 1;

    double M              = k;
    double N              = n * out_h * out_w;
    double K              = c * r * s;
    double tmp            = n * out_h * out_w * 1e-9f;
    double flops_per_conv = tmp * M * K * 2.0;

    float* input       = (float*)malloc(n * c * h * w * sizeof(float));
    float* weight      = (float*)malloc(k * c * r * s * sizeof(float));
    float* bias        = (float*)malloc(k * sizeof(float));
    float* output      = (float*)malloc(n * k * out_h * out_w * sizeof(float));
    float* output_host = (float*)malloc(n * k * out_h * out_w * sizeof(float));

    float *input_device, *weight_device, *bias_device, *output_device;
    hipMalloc((void**)&input_device, n * c * h * w * sizeof(float));
    hipMalloc((void**)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void**)&bias_device, k * sizeof(float));
    hipMalloc((void**)&output_device, n * k * out_h * out_w * sizeof(float));

    for(int i = 0; i < n * c * h * w; i++)
    {
        input[i] = (rand() % 255) / 255.0;
    }

    for(int i = 0; i < k * c * r * s; i++)
    {
        weight[i] = (rand() % 255) / 255.0;
    }

    for(int i = 0; i < k; i++)
    {
        bias[i] = 0.0f;
    }

    for(int i = 0; i < n * k * out_h * out_w; i++)
    {
        output[i]      = 0.0;
        output_host[i] = 0.0;
    }

    hipMemcpy(input_device, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_device, bias, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(
        output_device, output, n * k * out_h * out_w * sizeof(float), hipMemcpyHostToDevice);

    param_t param;

    param.input  = input_device;
    param.weight = weight_device;
    param.bias   = bias_device;
    param.output = output_device;
    param.n      = n;
    param.c      = c;
    param.h      = h;
    param.w      = w;
    param.k      = k;
    param.r      = r;
    param.s      = s;
    param.u      = u;
    param.v      = v;
    param.p      = p;
    param.q      = q;
    param.Oh     = out_h;
    param.Ow     = out_w;

    launch_implgemm(param);
    hipMemcpy(
        output_host, output_device, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float time_elapsed = 0.0;

    int iternum = 10;
    for(int i = 0; i < iternum; i++)
    {
        launch_implgemm(param);
    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("===================start verfiy===================\n");
    direct_conv2dcpu(input, weight, bias, output, n, c, h, w, k, r, s, u, v, p, q);

    int error = 0;
    for(int i = 0; i < n * k * out_h * out_w; i++)
    {
        if(abs(output_host[i] - output[i]) > getPrecision(output[i]))
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, output_host[i], output[i]);
            error++;
            break;
        }
    }
    printf("================finish,error:%d=========================\n", error);

    float timePerConv = time_elapsed / iternum;
    double gflops     = flops_per_conv / (timePerConv / 1000.0f);
    printf("%2d %2d %2d %2d %d %d %2d\n", n, h, w, c, r, s, k);
    printf("time: %f ms\n", timePerConv);
    printf("Performance :%f GFlops\n", gflops);

    hipFree(input_device);
    hipFree(weight_device);
    hipFree(output_device);

    free(input);
    free(weight);
    free(output);
    free(output_host);

    return 0;
}