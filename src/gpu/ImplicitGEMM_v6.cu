#include "hip/hip_runtime.h"
#include <cstdint>
#include <cstdlib>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "verify.h"
#include "conv_2d.h"

#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])

__global__ void implgemm(param_t param)
{
    __shared__ __align__(16 * 1024) char smem[24 * 1024];
    float* smem_weight = reinterpret_cast<float*>(smem);
    float* smem_input  = reinterpret_cast<float*>(smem + 16 * 1024);

    int tx = threadIdx.x;
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // warp tile
    const int lane_id   = tx % 32;
    const int warp_id   = tx / 32;
    const int mma_tid_x = lane_id % 8;
    const int mma_tid_y = lane_id / 8;

    // lds addr
    int weight_lds_addr = (warp_id / 2) * 32 + mma_tid_y * 4;
    int input_lds_addr  = (warp_id % 2) * 64 + mma_tid_x * 4;

    int z = blockIdx.z;

    float weight_ldg_reg[4];
    float input_ldg_reg[4];

    int pos_h[4];
    int pos_w[4];

#pragma unroll
    for(int i = 0; i < 4; i++)
    {
        pos_h[i] = ((bx * 128 + tx % 32 + i * 32) / param.Ow) * param.u - param.p;
        pos_w[i] = ((bx * 128 + tx % 32 + i * 32) % param.Ow) * param.v - param.q;
    }

    int input_offset         = z * param.c * param.h * param.w;
    int weight_offset        = (by * 128 + tx / 8 * 4) * param.c * param.r * param.s;
    int input_channel_offset = param.h * param.w;
    int weight_k_offset      = param.c * param.r * param.s;

    // sts addr
    int weight_sts_addr = (tx % 8) * 132 + (tx / 8) * 4;
    int input_sts_addr  = (tx / 32) * 128 + (tx % 32);

    int write_flag = 1;
    float weight_frag[2][8];
    float input_frag[2][8];
    float output_frag[8][8];
#pragma unroll
    for(int i = 0; i < 8; i++)
    {
#pragma unroll
        for(int j = 0; j < 8; j++)
        {
            output_frag[i][j] = 0.0f;
        }
    }

// ldg
#pragma unroll
    for(int i = 0; i < 4; i++)
    {
        if(tx % 8 < weight_k_offset && by * 128 + tx / 8 * 4 + i < param.k)
        {
            weight_ldg_reg[i] = param.weight[weight_offset + i * weight_k_offset + tx % 8];
        }
        else
        {
            weight_ldg_reg[i] = 0.0f;
        }
    }
    int cur_c = (tx / 32) / (param.r * param.s);
    int cur_r = ((tx / 32) % (param.r * param.s)) / param.s;
    int cur_s = ((tx / 32) % (param.r * param.s)) % param.s;
#pragma unroll
    for(int i = 0; i < 4; i++)
    {
        int cur_h            = pos_h[i] + cur_r;
        int cur_w            = pos_w[i] + cur_s;
        int input_offset_tmp = cur_c * input_channel_offset + cur_h * param.w + cur_w;
        if(cur_h >= 0 && cur_w >= 0 && cur_h < param.h && cur_w < param.w)
        {
            input_ldg_reg[i] = param.input[input_offset + input_offset_tmp];
        }
        else
        {
            input_ldg_reg[i] = 0.0f;
        }
    }
    // sts
    FLOAT4(smem_weight[weight_sts_addr]) = FLOAT4(weight_ldg_reg[0]);
#pragma unroll
    for(int i = 0; i < 4; i++)
    {
        smem_input[input_sts_addr + i * 32] = input_ldg_reg[i];
    }
    __syncthreads();

    // lds
    FLOAT4(weight_frag[0][0]) = FLOAT4(smem_weight[weight_lds_addr]);
    FLOAT4(weight_frag[0][4]) = FLOAT4(smem_weight[weight_lds_addr + 16]);

    FLOAT4(input_frag[0][0]) = FLOAT4(smem_input[input_lds_addr]);
    FLOAT4(input_frag[0][4]) = FLOAT4(smem_input[input_lds_addr + 32]);

    for(int crs = 0; crs < param.c * param.r * param.s; crs += 8)
    {
        // ldg
        int weight_offset_tmp = crs + 8 + tx % 8;
#pragma unroll
        for(int i = 0; i < 4; i++)
        {
            if(weight_offset_tmp < weight_k_offset && by * 128 + tx / 8 * 4 + i < param.k)
            {
                weight_ldg_reg[i] =
                    param.weight[weight_offset + weight_offset_tmp + i * weight_k_offset];
            }
            else
            {
                weight_ldg_reg[i] = 0.0f;
            }
        }
        cur_c = (tx / 32 + crs + 8) / (param.r * param.s);
        cur_r = ((tx / 32 + crs + 8) % (param.r * param.s)) / param.s;
        cur_s = ((tx / 32 + crs + 8) % (param.r * param.s)) % param.s;
#pragma unroll
        for(int i = 0; i < 4; i++)
        {
            int cur_h            = pos_h[i] + cur_r;
            int cur_w            = pos_w[i] + cur_s;
            int input_offset_tmp = cur_c * input_channel_offset + cur_h * param.w + cur_w;
            if(cur_h >= 0 && cur_w >= 0 && cur_h < param.h && cur_w < param.w)
            {
                input_ldg_reg[i] = param.input[input_offset + input_offset_tmp];
            }
            else
            {
                input_ldg_reg[i] = 0.0f;
            }
        }
        int load_flag = write_flag ^ 1; // 0
#pragma unroll
        for(int subcrs = 0; subcrs < 8 - 1; subcrs++)
        {
            // lds
            FLOAT4(weight_frag[(subcrs + 1) % 2][0]) =
                FLOAT4(smem_weight[load_flag * 132 * 8 + weight_lds_addr + (subcrs + 1) * 132]);
            FLOAT4(weight_frag[(subcrs + 1) % 2][4]) = FLOAT4(
                smem_weight[load_flag * 132 * 8 + weight_lds_addr + (subcrs + 1) * 132 + 16]);

            FLOAT4(input_frag[(subcrs + 1) % 2][0]) =
                FLOAT4(smem_input[load_flag * 128 * 8 + input_lds_addr + (subcrs + 1) * 128]);
            FLOAT4(input_frag[(subcrs + 1) % 2][4]) =
                FLOAT4(smem_input[load_flag * 128 * 8 + input_lds_addr + (subcrs + 1) * 128 + 32]);
// GEMM
#pragma unroll
            for(int i = 0; i < 8; i++)
            {
#pragma unroll
                for(int j = 0; j < 8; j++)
                {
                    output_frag[i][j] += weight_frag[subcrs % 2][i] * input_frag[subcrs % 2][j];
                }
            }
        }
        // sts
        FLOAT4(smem_weight[write_flag * 132 * 8 + weight_sts_addr]) = FLOAT4(weight_ldg_reg[0]);

#pragma unroll
        for(int i = 0; i < 4; i++)
        {
            smem_input[write_flag * 128 * 8 + input_sts_addr + i * 32] = input_ldg_reg[i];
        }
        __syncthreads();

        write_flag ^= 1; // 0
        // lds
        FLOAT4(weight_frag[0][0]) =
            FLOAT4(smem_weight[(load_flag ^ 1) * 132 * 8 + weight_lds_addr]);
        FLOAT4(weight_frag[0][4]) =
            FLOAT4(smem_weight[(load_flag ^ 1) * 132 * 8 + weight_lds_addr + 16]);

        FLOAT4(input_frag[0][0]) = FLOAT4(smem_input[(load_flag ^ 1) * 128 * 8 + input_lds_addr]);
        FLOAT4(input_frag[0][4]) =
            FLOAT4(smem_input[(load_flag ^ 1) * 128 * 8 + input_lds_addr + 32]);
        // GEMM
        for(int i = 0; i < 8; i++)
        {
            for(int j = 0; j < 8; j++)
            {
                output_frag[i][j] += weight_frag[1][i] * input_frag[1][j];
            }
        }
    }

    // reuse smem
    float* smem_output = reinterpret_cast<float*>(smem); // 4096
    float* smem_bias   = reinterpret_cast<float*>(smem + 16 * 1024);

    // bias ldg/sts
    if(tx < 128)
    {
        smem_bias[tx] = param.bias[by * 128 + tx];
    }

    uint32_t output_sts_addr = warp_id * 512 + mma_tid_y * 4 * 8 * 4 + mma_tid_x * 4;
    uint32_t output_lds_addr = warp_id * 512 + lane_id;

    uint32_t m_idx = by * 128 + (warp_id / 2) * 32;
    uint32_t n_idx = bx * 128 + (warp_id % 2) * 64 + lane_id;

#pragma unroll
    for(int i = 0; i < 2; i++)
    {
#pragma unroll
        for(int j = 0; j < 2; j++)
        {
            __syncthreads();
#pragma unroll
            for(int subi = 0; subi < 4; subi++)
            {
#pragma unroll
                for(int subj = 0; subj < 4; subj++)
                {
                    smem_output[output_sts_addr + subi * 8 * 4 + subj] =
                        output_frag[i * 4 + subi][j * 4 + subj];
                }
            }
            __syncthreads();

#pragma unroll
            for(int k = 0; k < 16; k++)
            {
                int out_offset = z * param.k * param.Oh * param.Ow +
                                 (m_idx + i * 16 + k) * param.Oh * param.Ow + n_idx + j * 32;
                if((m_idx + i * 16 + k) < param.k && (n_idx + j * 32) < param.Oh * param.Ow)
                {
                    param.output[out_offset] = smem_output[output_lds_addr + k * 32] +
                                               smem_bias[(warp_id / 2) * 32 + i * 16 + k];
                }
            }
        }
    }
}

void launch_implgemm(param_t param)
{
    unsigned int n = param.n;
    unsigned int h = param.h;
    unsigned int w = param.w;
    unsigned int k = param.k;
    unsigned int r = param.r;
    unsigned int s = param.s;
    unsigned int u = param.u;
    unsigned int v = param.v;
    unsigned int p = param.p;
    unsigned int q = param.q;

    int out_h = (h - r + 2 * p) / u + 1;
    int out_w = (w - s + 2 * q) / v + 1;

    param.Oh = out_h;
    param.Ow = out_w;

    int block_x  = ((out_h * out_w + 127) / 128);
    int block_y  = (k + 127) / 128;
    int block_z  = n;
    int thread_x = 256;
    int thread_y = 1;
    int thread_z = 1;

    dim3 grid(block_x, block_y, block_z);
    dim3 block(thread_x, thread_y, thread_z);

    implgemm<<<grid, block>>>(param);
}

int main(int argc, char** argv)
{
    unsigned int n = atoi(argv[1]);
    unsigned int c = atoi(argv[2]);
    unsigned int h = atoi(argv[3]);
    unsigned int w = atoi(argv[4]);
    unsigned int k = atoi(argv[5]);
    unsigned int r = atoi(argv[6]);
    unsigned int s = atoi(argv[7]);
    unsigned int u = atoi(argv[8]);
    unsigned int v = atoi(argv[9]);
    unsigned int p = atoi(argv[10]);
    unsigned int q = atoi(argv[11]);

    int out_h = (h - r + 2 * p) / u + 1;
    int out_w = (w - s + 2 * q) / v + 1;

    double M              = k;
    double N              = n * out_h * out_w;
    double K              = c * r * s;
    double tmp            = n * out_h * out_w * 1e-9f;
    double flops_per_conv = tmp * M * K * 2.0;

    float* input       = (float*)malloc(n * c * h * w * sizeof(float));
    float* weight      = (float*)malloc(k * c * r * s * sizeof(float));
    float* bias        = (float*)malloc(k * sizeof(float));
    float* output      = (float*)malloc(n * k * out_h * out_w * sizeof(float));
    float* output_host = (float*)malloc(n * k * out_h * out_w * sizeof(float));

    float *input_device, *weight_device, *bias_device, *output_device;
    hipMalloc((void**)&input_device, n * c * h * w * sizeof(float));
    hipMalloc((void**)&weight_device, k * c * r * s * sizeof(float));
    hipMalloc((void**)&bias_device, k * sizeof(float));
    hipMalloc((void**)&output_device, n * k * out_h * out_w * sizeof(float));

    for(int i = 0; i < n * c * h * w; i++)
    {
        input[i] = (rand() % 255) / 255.0;
    }

    for(int i = 0; i < k * c * r * s; i++)
    {
        weight[i] = (rand() % 255) / 255.0;
    }

    for(int i = 0; i < k; i++)
    {
        bias[i] = (rand() % 255) / 255.0;
    }

    for(int i = 0; i < n * k * out_h * out_w; i++)
    {
        output[i]      = 0.0;
        output_host[i] = 0.0;
    }

    hipMemcpy(input_device, input, n * c * h * w * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(weight_device, weight, k * c * r * s * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(bias_device, bias, k * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(
        output_device, output, n * k * out_h * out_w * sizeof(float), hipMemcpyHostToDevice);

    param_t param;

    param.input  = input_device;
    param.weight = weight_device;
    param.bias   = bias_device;
    param.output = output_device;
    param.n      = n;
    param.c      = c;
    param.h      = h;
    param.w      = w;
    param.k      = k;
    param.r      = r;
    param.s      = s;
    param.u      = u;
    param.v      = v;
    param.p      = p;
    param.q      = q;
    param.Oh     = out_h;
    param.Ow     = out_w;

    launch_implgemm(param);
    hipMemcpy(
        output_host, output_device, n * k * out_h * out_w * sizeof(float), hipMemcpyDeviceToHost);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    float time_elapsed = 0.0;

    int iternum = 10;
    for(int i = 0; i < iternum; i++)
    {
        launch_implgemm(param);
    }
    hipEventRecord(stop, 0);

    hipEventSynchronize(stop);
    hipEventElapsedTime(&time_elapsed, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("===================start verfiy===================\n");
    direct_conv2dcpu(input, weight, bias, output, n, c, h, w, k, r, s, u, v, p, q);

    int error = 0;
    for(int i = 0; i < n * k * out_h * out_w; i++)
    {
        if(abs(output_host[i] - output[i]) > getPrecision(output[i]))
        {
            printf("error, postion:%d, gpuvalue:%f, cpuvalue:%f\n", i, output_host[i], output[i]);
            error++;
            break;
        }
    }
    printf("================finish,error:%d=========================\n", error);

    float timePerConv = time_elapsed / iternum;
    double gflops     = flops_per_conv / (timePerConv / 1000.0f);
    printf("%2d %2d %2d %2d %d %d %2d\n", n, h, w, c, r, s, k);
    printf("time: %f ms\n", timePerConv);
    printf("Performance :%f GFlops\n", gflops);

    hipFree(input_device);
    hipFree(weight_device);
    hipFree(output_device);

    free(input);
    free(weight);
    free(output);
    free(output_host);

    return 0;
}